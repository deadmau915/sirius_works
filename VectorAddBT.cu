#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

#define SIZE 2048

__global__ void VectorAdd(int *a, int *b, int *c)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i<SIZE)
		c[i]=a[i]+b[i];
}

int main()
{

	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	
	a = (int *)malloc(SIZE*sizeof(int));
	b = (int *)malloc(SIZE*sizeof(int));
	c = (int *)malloc(SIZE*sizeof(int));
	
	hipMalloc(&d_a, SIZE*sizeof(int));
	hipMalloc(&d_b, SIZE*sizeof(int));
	hipMalloc(&d_c, SIZE*sizeof(int));

	for(int i=0;i<SIZE;i++)
	{
		a[i]=i;
		b[i]=i;
		c[i]=0;	
	}

	hipMemcpy(d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice);

	dim3 dimblock(1024,1,1);
	dim3 dimGrid(ceil(SIZE/1024.0),1,1);

	VectorAdd<<< dimGrid,dimblock >>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost);

	for(int i=0;i<10; i++)
		printf("%d ",c[i]);
		
	free(a);
	free(b);
	free(c);
	
	hipFree(d_a);	
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
